
#include <hip/hip_runtime.h>
#include "stdio.h" 
#include "stdlib.h"
#define MAX_NC 32
#define ERROR_HANDLING(call) {														\
    hipError_t error = call;												\
    if(error != hipSuccess) {											\
        fprintf(stderr, "ERROR: in file '%s' in line %i: %s.\n",	\
            __FILE__, __LINE__, hipGetErrorString(error));				\
        exit(1);														\
    }																	\
} while (0)


typedef struct double24 {double x; double y; double z;} double24;

__device__
double24 rgb_sub(uchar4 *a, double24 *b) {
    double24 res = {a->x - b->x, a->y - b->y, a->z - b->z};
    return res;
}
__device__
double rgb_mul(double24 *a, double24 *b) {
    return a->x * b->x + a->y * b->y + a->z * b->z;
}

__constant__ double24 cavg_const[MAX_NC];
__constant__ int nc_const;

void copy_from_ram2dev_const(double24* cavg, int *nc) {
    ERROR_HANDLING(hipMemcpyToSymbol(HIP_SYMBOL(nc_const), nc, sizeof(int)));
    ERROR_HANDLING(hipMemcpyToSymbol(HIP_SYMBOL(cavg_const), cavg, (*nc)*sizeof(double24)));
}

__global__
void min_dist_method_kernel(uchar4* image, size_t width, size_t height) {
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t idy = threadIdx.y + blockIdx.y * blockDim.y;
    size_t offsetx = blockDim.x * gridDim.x;
    size_t offsety = blockDim.y * gridDim.y;
    for (size_t x = idx; x < width; x+=offsetx) {
        for (size_t y = idy; y < height; y+=offsety) {
            uchar4* pixel = &image[x*height + y];
            
            int cur_max_class_idx = 0;
            double24 sub_res = rgb_sub(pixel, &cavg_const[cur_max_class_idx]);
            double cur_max_value = -rgb_mul(&sub_res, &sub_res);
            
            for (int i = 1; i < nc_const; i++) {
                double24 sub_res = rgb_sub(pixel, &cavg_const[i]);
                double res = - rgb_mul(&sub_res, &sub_res);
                if (res > cur_max_value) {
                    cur_max_value = res;
                    cur_max_class_idx = i;
                }
            }
            pixel->w  = cur_max_class_idx;
        }
    }
}

int main() {
    int width, height;
    char inputFilename[256];
    char outputFilename[256];
    scanf("%s", inputFilename);
    scanf("%s", outputFilename);
    FILE* file = fopen(inputFilename, "rb");

    fread(&width, sizeof(int), 1, file);
    fread(&height, sizeof(int), 1, file);
    uchar4* image = (uchar4*)malloc(sizeof(uchar4)*height*width);
    fread(image, sizeof(uchar4), width*height, file);
    fclose(file);

    int nc;
    scanf("%d", &nc);

    double24 cavg[MAX_NC];

    int np_i;
    for (int i = 0; i < nc; i++) {
    	scanf("%d", &np_i);
        double24 tmp_sum = {0., 0., 0.};
        int x, y;
    	for (int j = 0; j < np_i; j++) {
    		scanf("%d %d", &x, &y);
            tmp_sum.x += image[y*width+x].x;
            tmp_sum.y += image[y*width+x].y;
            tmp_sum.z += image[y*width+x].z;
    	}
        cavg[i].x = tmp_sum.x/np_i;
        cavg[i].y = tmp_sum.y/np_i;
        cavg[i].z = tmp_sum.z/np_i;
    }
    copy_from_ram2dev_const(cavg, &nc);	
    uchar4* dimage;
    ERROR_HANDLING(hipMalloc((void**)&dimage, sizeof(uchar4)*width*height));
    ERROR_HANDLING(hipMemcpy(dimage, image, sizeof(uchar4)*width*height, hipMemcpyHostToDevice));
    min_dist_method_kernel<<<dim3(16, 16), dim3(16, 16)>>>(dimage, width, height);
    ERROR_HANDLING(hipGetLastError());
    ERROR_HANDLING(hipMemcpy(image, dimage, sizeof(uchar4)*width*height, hipMemcpyDeviceToHost));
    ERROR_HANDLING(hipFree(dimage));
    file = fopen(outputFilename, "wb");
    fwrite(&width, sizeof(int), 1, file);
    fwrite(&height, sizeof(int), 1, file);
    fwrite(image, sizeof(uchar4), width * height, file);
    fclose(file);
    free(image);
    return 0;
}
