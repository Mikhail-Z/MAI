#include "hip/hip_runtime.h"

#include <cstdio>
#include <cstdlib>


__global__ void kernelDeduct(double *a, double* b, double* c, size_t n) {
        size_t i = blockDim.x*blockIdx.x + threadIdx.x;
	size_t offset = gridDim.x*blockDim.x;
	for (; i < n; i+= offset){
		c[i] = a[i] - b[i];
	}
}


int main() {

        size_t n;

        scanf("%lu\n", &n);

        double *a = (double *)malloc(sizeof(double)*n);
        double *b = (double *)malloc(sizeof(double)*n);
        double *c = (double *)malloc(sizeof(double)*n);
	
	for (size_t i = 0; i < n; i++)
        	scanf("%lf", &a[i]);
	for (size_t i = 0; i < n; i++)
        	scanf("%lf", &b[i]);

        hipError_t cudaStatus = hipSetDevice(0);

        if (cudaStatus != hipSuccess) {
	    printf("ERROR: %s\n", "hipSetDevice check failed. You must have at least one Nvidia GPU!");
	    return 0;
	}
        double *va, *vb, *vc;


        cudaStatus = hipMalloc((void**)&va, n*sizeof(double));
	if (cudaStatus != hipSuccess) {
		printf("ERROR: %s\n", "Can't allocate video memory");
		return 0;
	}

        cudaStatus = hipMalloc((void**)&vb, n*sizeof(double));
	if (cudaStatus != hipSuccess) {
		printf("ERROR: %s\n", "Can't allocate video memory");
		return 0;
	}

        cudaStatus = hipMalloc((void**)&vc, n*sizeof(double));
	if (cudaStatus != hipSuccess) {
		printf("ERROR: %s\n", "Can't allocate video memory");
		return 0;
	}


        cudaStatus = hipMemcpy(va, a, n*sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("ERROR: %s\n", "Can't copy from ram to videomemory");
		return 0;
	}

        cudaStatus = hipMemcpy(vb, b, n*sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("ERROR: %s\n", "Can't copy from ram to videomemory");
		return 0;
	}

        kernelDeduct <<<16384, 512>>>(va, vb, vc, n);

	cudaStatus = hipMemcpy(c, vc, n*sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("ERROR: %s\n", "Can't copy from videomemory to ram");
		return 0;
	}

        hipFree(va);
        hipFree(vb);
        hipFree(vc);

	for (size_t i = 0; i < n; i++)
            printf("%.10e ", c[i]);
	printf("\n");
        return 0;

}

