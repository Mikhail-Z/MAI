#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "stdio.h"
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>
#include "math.h"
#define BLOCK_SIZE 32

__global__
void swap_cols_kernel(double *a, int col1_idx, int col2_idx, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    for (int row = idx; row < n; row+=offset) {
        double tmp = a[n*idx+col1_idx];
        a[n*idx+col1_idx] = a[n*idx+col2_idx];
        a[n*idx+col2_idx] = tmp;
    }
}

__global__
void coef_mul_and_sub_kernel(double *a, int fst_col_idx, int fst_row_idx, double *coefs, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offset_x = gridDim.x * blockDim.x;
    int offset_y = gridDim.y * blockDim.y;
    for (int i = fst_col_idx + idx; i < n; i+=offset_x) {
        for (int j = fst_row_idx+idy; j < n; j+=offset_y) {
            int diag_elem_idx = n*j+fst_col_idx-1;
            a[n*j + i] -=coefs[i]*a[diag_elem_idx];
        }

    }
}

struct compare_abs_value
{
    __host__ __device__
    bool operator()(double a, double b) {
        return (a<0? -a:a) < ( b<0? -b:b);
    }

};


void print_matrix(double *a, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++)
            printf("%.10e ", a[j*n+i]);
        printf("\n");
    }
}

__global__
void set_cur_row_elements_kernel(double* a, double* coefs, int n, int row, int fst_col_idx) {
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    int offset = blockDim.x*gridDim.x;
    for (int col = idx+fst_col_idx; col < n; col+=offset) {
        coefs[col] = a[n*row+col]/a[n*row+fst_col_idx-1];
        a[n*row+col] = coefs[col];
    }

}


int main() {
    int n;
    scanf("%d", &n);
    double* a = (double*)malloc(sizeof(double)*n*n);
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++)
            scanf("%lf", &a[j*n+i]);
    }
    int size = sizeof(double)*n*n;
    double* d_a;
    hipMalloc(&d_a, size);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    int* swap_vector = (int*)malloc(sizeof(int)*n);
    double *d_coefs;
    double *coefs = (double*)malloc(sizeof(double)*n);
    hipMalloc(&d_coefs, n*sizeof(double));
    for (int row = 0; row < n-1; row++) {
        thrust::device_ptr<double> d_ptr = thrust::device_pointer_cast(&d_a[row*n+row]);

        thrust::device_ptr<double> d_row_begin_ptr = thrust::device_pointer_cast(&d_a[row*n]);
        thrust::device_ptr<double> max_elem_ptr = thrust::max_element(d_ptr, d_row_begin_ptr + n, compare_abs_value());
        int max_elem_idx = max_elem_ptr - d_row_begin_ptr;

        swap_vector[row] = max_elem_idx;

        swap_cols_kernel<<<(n+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_a, row, max_elem_idx, n);

        set_cur_row_elements_kernel<<<((n-row-1)+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_a, d_coefs, n, row, row+1);
        hipMemcpy(coefs, d_coefs, sizeof(double)*n, hipMemcpyDeviceToHost);


        dim3 dimGrid = dim3(((n-row-1)+BLOCK_SIZE-1)/BLOCK_SIZE, ((n-row-1)+BLOCK_SIZE-1)/BLOCK_SIZE);
        dim3 dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
        coef_mul_and_sub_kernel<<<dimGrid, dimBlock>>>(d_a, row+1, row+1,  d_coefs, n);

    }
    swap_vector[n-1] = n-1;
    hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
    print_matrix(a, n);

    int* p = (int*)malloc(sizeof(int)*n);
    for (int i = 0; i < n; i++)
        p[i] = i;
    for (int i = n-1; i >= 0; i--) {
        int tmp = p[i];
        p[i] = p[swap_vector[i]];
        p[swap_vector[i]] = tmp;
    }
    for (int i = 0; i < n; i++)
        printf("%d ", p[i]);

    hipFree(d_a);
    hipFree(coefs);
    free(a);
    free(p);
    free(swap_vector);
    
    return 0;
}
