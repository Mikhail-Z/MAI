#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include <cmath>
using namespace std;
#define ERROR_HANDLING(call) {														\
    hipError_t error = call;												\
    if(error != hipSuccess) {											\
        fprintf(stderr, "ERROR: in file '%s' in line %i: %s.\n",	\
            __FILE__, __LINE__, hipGetErrorString(error));				\
        exit(1);														\
    }																	\
} while (0)

texture<uchar4, 2, hipReadModeElementType> texRef2D;

__device__
double getBrightnessFromRGB(uchar4 p) {
	return 0.299*p.x + 0.587*p.y + 0.114*p.z;
}

__device__
int convolution(double window[3][3]) {
	double Gx = window[0][2] - window[0][0] + window[1][2] - window[1][0] + window[2][2] - window[2][0];
	double Gy = window[2][0] - window[0][0] + window[2][1] - window[0][1] + window[2][2] - window[0][2];

	double eps = 1e-10;
	double G = sqrt(Gx*Gx+Gy*Gy);
	int res =  abs(((int)G+1.0)-G)<eps ? (int)G+1 : (int)G;
	if (res < 0)
		res = 0;
	else if (res > 255)
		res = 255;
	return res;
}


__global__
void PrewittKernel(uchar4 *device_data, int width, int height) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int x, y;
    const int windowWidth = 3;
    const int windowHeight = 3;
	for(x = idx; x < width; x += offsetx) 
		for(y = idy; y < height; y += offsety) {
			uchar4 windowPoints[windowHeight][windowWidth];
			uchar4 centerPoint = tex2D(texRef2D, x, y);
			windowPoints[0][0] = tex2D(texRef2D, x-1, y-1);
			windowPoints[0][1] = tex2D(texRef2D, x, y-1);
			windowPoints[0][2] = tex2D(texRef2D, x+1, y-1);
			windowPoints[1][0] = tex2D(texRef2D, x-1, y);
			windowPoints[1][1] = centerPoint;
			windowPoints[1][2] = tex2D(texRef2D, x+1, y);
			windowPoints[2][0] = tex2D(texRef2D, x-1, y+1);
			windowPoints[2][1] = tex2D(texRef2D, x, y+1);
			windowPoints[2][2] = tex2D(texRef2D, x+1, y+1);
			double windowBrightnesses[windowWidth][windowHeight];
			for (int i = 0; i < windowHeight; i++)
				for (int j = 0; j < windowWidth; j++) {
					windowBrightnesses[i][j] = getBrightnessFromRGB(windowPoints[i][j]);
			}

			int resBrightness = convolution(windowBrightnesses);
			uchar4 resPoint = make_uchar4(resBrightness, resBrightness, resBrightness, centerPoint.w);
			device_data[y * width + x] = resPoint;
		}
}

int main(int argc, char *argv[]) {
	int width, height;
    char inputFilename[256];
    char outputFilename[256];
    scanf("%s", inputFilename);
    scanf("%s", outputFilename);	
    FILE* file = fopen(inputFilename, "rb");

	fread(&width, sizeof(int), 1, file);
	fread(&height, sizeof(int), 1, file);
	uchar4* io_data = (uchar4*)malloc(sizeof(uchar4)*height*width);
	
	fread(io_data, sizeof(uchar4), width*height, file);
	fclose(file);

	hipArray *c_arr;
	hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
	ERROR_HANDLING(hipMallocArray(&c_arr, &ch, width, height));
	ERROR_HANDLING(hipMemcpyToArray(c_arr, 0, 0, io_data, sizeof(uchar4) * width * height, hipMemcpyHostToDevice));

	texRef2D.addressMode[0] = hipAddressModeClamp;
	texRef2D.addressMode[1] = hipAddressModeClamp;
	texRef2D.channelDesc = ch;
	texRef2D.filterMode = hipFilterModePoint;
	texRef2D.normalized = false;


	ERROR_HANDLING(hipBindTextureToArray(texRef2D, c_arr, ch));
	
	uchar4 *dev_data;
	ERROR_HANDLING(hipMalloc(&dev_data, sizeof(uchar4) * width * height));

	PrewittKernel<<<dim3(32, 32), dim3(16, 16)>>>(dev_data, width, height);
	ERROR_HANDLING(hipGetLastError());

	ERROR_HANDLING(hipMemcpy(io_data, dev_data, sizeof(uchar4) * width * height, hipMemcpyDeviceToHost));
	ERROR_HANDLING(hipUnbindTexture(texRef2D));
	ERROR_HANDLING(hipFreeArray(c_arr));
	ERROR_HANDLING(hipFree(dev_data));

	file = fopen(outputFilename, "wb");
	fwrite(&width, sizeof(int), 1, file);
	fwrite(&height, sizeof(int), 1, file);
	fwrite(io_data, sizeof(uchar4), width * height, file);
	fclose(file);
    free(io_data);
	return 0;
} 
