
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"

#define SHARED_MEMORY_MAX_SIZE 1024
#define UINT_SIZE 32

#define BANKS_NUM 32
#define LOG_BANKS_NUM 5

#define CONFLICT_FREE_OFFSET(n) \
    (n >>  + n >> (2 * LOG_BANKS_NUM))

#define ERROR_HANDLING(call) {                                                                                                          \
    hipError_t error = call;                                                                                             \
    if(error != hipSuccess) {                                                                                  \
        fprintf(stderr, "ERROR: file '%s' in line %i: %s.\n",   \
            __FILE__, __LINE__, hipGetErrorString(error));                             \
        exit(1);                                                                                                                \
    }                                                                                                                                   \
} while (0)


__device__ __host__
void print_array(int* a, int n) {
    for (int i = 0; i < n; i++)
        printf("%d ", a[i]);
    printf("\n");
}


__global__
void scan(int *a, int *s, int n, int all_n)
{
    extern __shared__ int temp[];

    int th_id = threadIdx.x;
    int th_offset = 2*(blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x);
    int offset = 1;
    int leftIdx = th_id;
    int rightIdx = th_id + (n/2);

    int bankLeftOffset = CONFLICT_FREE_OFFSET(leftIdx);
    int bankRightOffset = CONFLICT_FREE_OFFSET(rightIdx);

    if (th_offset + th_id+(n/2) < all_n+1) {
        temp[leftIdx + bankLeftOffset] = a[th_offset+leftIdx];
        temp[rightIdx + bankRightOffset] = a[th_offset+rightIdx];

        for (int d = n/2; d > 0; d /= 2)
        {
            __syncthreads();
            if (th_id < d)
            {
                int leftIdx = offset*(2*th_id+1)-1;
                int rightIdx = offset*(2*th_id+2)-1;
                leftIdx += CONFLICT_FREE_OFFSET(leftIdx);
                rightIdx += CONFLICT_FREE_OFFSET(rightIdx);

                temp[rightIdx] += temp[leftIdx];
            }
            offset *= 2;
        }
        if (th_id==0)
            temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;

        for (int d = 1; d < n; d *= 2)
        {
            offset /= 2;
            __syncthreads();
            if (th_id < d)
            {
                int leftIdx = offset*(2*th_id+1)-1;
                int rightIdx = offset*(2*th_id+2)-1;
                leftIdx += CONFLICT_FREE_OFFSET(leftIdx);
                rightIdx += CONFLICT_FREE_OFFSET(rightIdx);

                int tmp = temp[leftIdx];
                temp[leftIdx] = temp[rightIdx];
                temp[rightIdx] += tmp;
            }
        }
        __syncthreads();

        s[th_offset+rightIdx] = temp[rightIdx + bankRightOffset];
        if (th_id != 0)
            s[th_offset+leftIdx] = temp[leftIdx + bankLeftOffset];

        if (th_id == ((n/2)-1))
            s[th_offset+rightIdx+1] = s[th_offset+rightIdx] + a[th_offset+rightIdx];
    }
}


__device__ __host__
int min2(int a, int b) {
    return a < b? a:b;
}


__global__
void get_last_elements(int* a, int* last_elements, int n, int block_size, int a_len) {
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    int offset = blockDim.x * gridDim.x;
    for (int i = idx; i < n; i+=offset) {
        last_elements[i] = a[min2(i * block_size + block_size, a_len)];
    }
}

__global__
void sub(int* a, int* b, int n) {
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    int offset = blockDim.x * gridDim.x;
    for (int i = idx; i < n; i+=offset)
        a[i] = a[i] - b[i];
}

__global__
void sum(int* s, int* diff, int n) {
    int blockIdx2sum = blockIdx.y*gridDim.x + blockIdx.x;
    int thIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
    if (thIdx < n)
        s[thIdx] += diff[blockIdx2sum];
}

void full_scan(int* a, int* s, int n) {
    if (n > SHARED_MEMORY_MAX_SIZE) {
        int blocks_count = (n+SHARED_MEMORY_MAX_SIZE-1)/SHARED_MEMORY_MAX_SIZE;
        int max_blocks_per_axis = 32768;
        if ((n % SHARED_MEMORY_MAX_SIZE) != 0) {
            int* tmp_s, *tmp_a;
            int* last_block_elements;
            ERROR_HANDLING(hipMalloc(&last_block_elements, sizeof(n)*blocks_count));
            int threadblocks_count_y = blocks_count % max_blocks_per_axis == 0? blocks_count/max_blocks_per_axis : blocks_count/max_blocks_per_axis+1;
            int threadblocks_count_x = blocks_count <= max_blocks_per_axis? blocks_count : max_blocks_per_axis;

            dim3 dimGrid(threadblocks_count_x, threadblocks_count_y);
            int new_n = n + (SHARED_MEMORY_MAX_SIZE - n % SHARED_MEMORY_MAX_SIZE);
            size_t new_size = sizeof(int)*new_n;
            ERROR_HANDLING(hipMalloc(&tmp_a, new_size));
            ERROR_HANDLING(hipMalloc(&tmp_s, new_size+sizeof(int)));
            ERROR_HANDLING(hipMemcpy(tmp_a, a, n*sizeof(int), hipMemcpyDeviceToDevice));
            ERROR_HANDLING(hipMemset(tmp_a+n, 0, sizeof(int)*(SHARED_MEMORY_MAX_SIZE - n % SHARED_MEMORY_MAX_SIZE)));
            ERROR_HANDLING(hipMemset(tmp_s, 0, new_size+sizeof(int)));
	    ERROR_HANDLING(hipGetLastError());
            scan <<<dimGrid, SHARED_MEMORY_MAX_SIZE/2, SHARED_MEMORY_MAX_SIZE*sizeof(int)>>>(tmp_a, tmp_s, SHARED_MEMORY_MAX_SIZE, new_n);
            ERROR_HANDLING(hipGetLastError());

            ERROR_HANDLING(hipMemcpy(s, tmp_s, (n+1)*sizeof(int), hipMemcpyDeviceToDevice));
            ERROR_HANDLING(hipFree(tmp_a));
            ERROR_HANDLING(hipFree(tmp_s));
        }
        else {
            int threadblocks_count_y = blocks_count % max_blocks_per_axis == 0? blocks_count/max_blocks_per_axis : blocks_count/max_blocks_per_axis+1;
            int threadblocks_count_x = blocks_count <= max_blocks_per_axis? blocks_count : max_blocks_per_axis;
            dim3 dimGrid(
                    threadblocks_count_x,
                    threadblocks_count_y);
            scan <<< dimGrid, SHARED_MEMORY_MAX_SIZE/2, SHARED_MEMORY_MAX_SIZE*sizeof(int)>>>(a, s, SHARED_MEMORY_MAX_SIZE, n);
            ERROR_HANDLING(hipGetLastError());
        }

        int* last_block_elements;
        ERROR_HANDLING(hipMalloc(&last_block_elements, sizeof(n)*blocks_count));
        get_last_elements<<<1024, 256>>>(s, last_block_elements, blocks_count, SHARED_MEMORY_MAX_SIZE, n);
        ERROR_HANDLING(hipGetLastError());
        int* new_s;
        ERROR_HANDLING(hipMalloc(&new_s, sizeof(int)*(blocks_count+1)));
        ERROR_HANDLING(hipMemset(new_s, 0, sizeof(int)*(blocks_count+1)));

        full_scan(last_block_elements, new_s, blocks_count);

        sub<<<1024, 256>>>(new_s+1, last_block_elements, blocks_count);
        ERROR_HANDLING(hipGetLastError());

        int threadblocks_count_y = blocks_count % max_blocks_per_axis == 0? blocks_count/max_blocks_per_axis : blocks_count/max_blocks_per_axis+1;
        int threadblocks_count_x = blocks_count <= max_blocks_per_axis? blocks_count : max_blocks_per_axis;

        dim3 dimGrid(threadblocks_count_x, threadblocks_count_y);

        sum<<<dimGrid, SHARED_MEMORY_MAX_SIZE>>>(s+1, new_s+1, n);
        ERROR_HANDLING(hipGetLastError());
    }
    else {
        dim3 dimGrid(1, 1);
        if ((n % SHARED_MEMORY_MAX_SIZE) != 0) {
            int* tmp_s, *tmp_a;
            size_t new_size = sizeof(int)*SHARED_MEMORY_MAX_SIZE;
            ERROR_HANDLING(hipMalloc(&tmp_a, new_size));
            ERROR_HANDLING(hipMalloc(&tmp_s, new_size+sizeof(int)));
            ERROR_HANDLING(hipMemcpy(tmp_a, a, n*sizeof(int), hipMemcpyDeviceToDevice));
            ERROR_HANDLING(hipMemset(tmp_a+n, 0, sizeof(int)*(SHARED_MEMORY_MAX_SIZE - n % SHARED_MEMORY_MAX_SIZE)));
            ERROR_HANDLING(hipMemset(tmp_s, 0, sizeof(int)*(SHARED_MEMORY_MAX_SIZE+1)));

            scan <<<dimGrid, SHARED_MEMORY_MAX_SIZE/2, new_size>>>(tmp_a, tmp_s, SHARED_MEMORY_MAX_SIZE, SHARED_MEMORY_MAX_SIZE);
            ERROR_HANDLING(hipGetLastError());
            hipMemcpy(s+1, tmp_s+1, n*sizeof(int), hipMemcpyDeviceToDevice);
            hipFree(tmp_a);
            //cudaFree(tmp_s);
        }
        else {
            scan <<<dimGrid, SHARED_MEMORY_MAX_SIZE/2, n*sizeof(int)>>>(a, s, n, n);
            ERROR_HANDLING(hipGetLastError());
        }
    }
}


__global__
void get_digit(uint *a, int *a_digits, int digit_number, int n) {
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    int offset = blockDim.x*gridDim.x;
    for (int i = idx; i < n; i+=offset) {
        a_digits[i] = (a[i]>>digit_number) & 1;
    }
}

__global__
void set_new_position(uint *a_out, uint *a_in, int *s, int *b, int n) {
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    int offset = blockDim.x*gridDim.x;
    for (int i = idx; i < n; i+=offset) {
        int new_pos = (b[i] == 0? i-s[i]: s[i] + n-s[n]);
        a_out[new_pos] = a_in[i];
    }
}


int main() {
    int n;
    fread(&n, sizeof(int), 1, stdin);

    int size = n*sizeof(int);

    uint* a = (uint*)malloc(size);
    fread(a, sizeof(uint), n, stdin);


    int* ds;
    uint *da;
    uint* da2;
    int *da_digits;
    hipMalloc(&ds, size+sizeof(int));
    hipMalloc(&da, size);
    hipMalloc(&da_digits, size);
    
    hipMemcpy(da, a, size, hipMemcpyHostToDevice);

    for (int digit_number = 0; digit_number < UINT_SIZE; digit_number++) {
        get_digit <<<1024, 256>>>(da, da_digits, digit_number, n);
	hipFree(da);
        hipMemset(ds, 0, size+sizeof(int));
        full_scan(da_digits, ds, n);
        ERROR_HANDLING(hipGetLastError());
	ERROR_HANDLING(hipMalloc(&da, size));
        ERROR_HANDLING(hipMalloc(&da2, size));
        ERROR_HANDLING(hipMemcpy(da, a, size, hipMemcpyHostToDevice));
        ERROR_HANDLING(hipMemcpy(da2, da, size, hipMemcpyDeviceToDevice));
        
        set_new_position <<<1024, 256>>>(da, da2, ds, da_digits, n);
	ERROR_HANDLING(hipMemcpy(a, da, size, hipMemcpyDeviceToHost));
        hipFree(da2);
    }
    //cudaMemcpy(a, da, size, cudaMemcpyDeviceToHost);
    fwrite(a, sizeof(uint), n, stdout);

    hipFree(da);
    hipFree(ds);
    hipFree(da_digits);


    free(a);
    return 0;
}
