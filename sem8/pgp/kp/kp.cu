#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/execution_policy.h>

#define CSC(call) {                         \
    hipError_t err = call;                       \
    if(err != hipSuccess) {                        \
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",    \
            __FILE__, __LINE__, hipGetErrorString(err));       \
        exit(1);                            \
    }                                   \
} while (0)


const int width = 1024;
const int height = 768;
const int points_num = 600;

float a1 = 0.1;
float a2 = 0.3;
int seed = 0;

const float dt = 0.02;
float inertion = 0.8;
const float pointRadius = 0.3;

const float fminPointRadius = 0.5;
float xCenter = 0.0, yCenter = 0.0, xScale = 100.0, yScale = xScale * height / width, minFvalue = 0;

float2 *dev_points;
float2 *dev_velocity, *dev_localBest;
float *dev_Fvalue;
float* dev_dist;
float err;
GLuint vbo;

__constant__ float dev_xCenter, dev_yCenter, dev_xScale, dev_yScale, dev_minFvalue, dev_maxFvalue;
__constant__ float2 dev_globalBest;

__constant__ int dev_seed;


__device__ __host__
float Rozenbrock(float x, float y) {
    return (1 - x) * (1 - x) + 100 * (y - x * x) * (y - x * x);
}

__global__
void get_dist_to_fmin_pos(float2* points, int n, float* dists) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int offset = gridDim.x*blockDim.x;
    for (int i = idx; i < n; i+=offset) {
        dists[i] = sqrtf(points[i].x*points[i].x + points[i].y*points[i].y);
    }
}

float get_mean_dist_to_fmin_pos() {
    get_dist_to_fmin_pos<<<32, 32>>>(dev_points, points_num, dev_dist);
    float mean_dist_to_fmin_pos = thrust::reduce(thrust::device, dev_dist, dev_dist + points_num)/points_num;
    CSC(hipGetLastError());
    return mean_dist_to_fmin_pos;
}


__device__ __host__
float Rozenbrock(float2 p) {
    return Rozenbrock(p.x, p.y);
}

struct cmp {
    __device__ bool operator()(float2 p1, float2 p2) {
        return Rozenbrock(p1) < Rozenbrock(p2);
    }
};

__device__ float2 operator+(float2 p1, float2 p2) {
return make_float2(p1.x + p2.x, p1.y + p2.y);
}

__device__ float2 transformCoordToPixel(float x, float y) {
    int col = ((x-dev_xCenter)/dev_xScale+1.0)*(width-1)/2;
    int row = ((y-dev_yCenter)/dev_yScale+1.0)*(height-1)/2;
    return make_float2(col, row);
}
__device__ float2 transformPixelToCoord(int row, int col) {
    float x = (float(2 * col)/(width - 1) - 1.0)*dev_xScale+dev_xCenter;
    float y = (float(2 * row)/(height - 1) - 1.0)*dev_yScale+dev_yCenter;
    return make_float2(x, y);
}

__global__ void calculateF(float *f_val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;
    int i, j;
    for (i = idx; i < width; i += offsetx) {
        for (j = idy; j < height; j += offsety) {
	    float2 coord = transformPixelToCoord(j, i);
            float f = Rozenbrock(coord);
            f_val[j * width + i] = f;
        }
    }
}

__host__ __device__
void print_array(float2* a, int n) {
    for (int i = 0; i < n; i++)
        printf("%.3f %.3f\n", a[i].x, a[i].y);
}

__global__ void updateCoord(float2* points, float2* v, int n) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int offset = blockDim.x*gridDim.x;
    for (int i = idx; i < n; i += offset) {
        points[i].x += v[i].x*dt;
        points[i].y += v[i].y*dt;
    }
}


__global__ void drawPoints(uchar4* data, float2* points, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    for (int i = idx; i < n; i += offset) {
        float2 _left_top_pixel = transformCoordToPixel(points[i].x-pointRadius, points[i].y-pointRadius);
        float2 _right_bottom_pixel = transformCoordToPixel(points[i].x+pointRadius, points[i].y+pointRadius);
        int2 right_bottom_pixel = make_int2(int(_right_bottom_pixel.x+0.5), int(_right_bottom_pixel.y+0.5));
        int2 left_top_pixel = make_int2(int(_left_top_pixel.x+0.5), int(_left_top_pixel.y+0.5));

        if (right_bottom_pixel.x - left_top_pixel.x == 0)
            right_bottom_pixel.x += 1;
        if (right_bottom_pixel.y - left_top_pixel.y == 0)
            right_bottom_pixel.y += 1;
        for (int i = max(0, left_top_pixel.x); i < min(width, right_bottom_pixel.x); i++) {
            for (int j = max(0,left_top_pixel.y); j < min(height, right_bottom_pixel.y); j++) {
                data[j * width + i] = make_uchar4(0, 0, 0, 255);
            }
        }
    }

    float2 _left_top_pixel = transformCoordToPixel(dev_globalBest.x-fminPointRadius, dev_globalBest.y-fminPointRadius);
    float2 _right_bottom_pixel = transformCoordToPixel(dev_globalBest.x+fminPointRadius, dev_globalBest.y+fminPointRadius);
    int2 right_bottom_pixel = make_int2(int(_right_bottom_pixel.x+0.5), int(_right_bottom_pixel.y+0.5));
    int2 left_top_pixel = make_int2(int(_left_top_pixel.x+0.5), int(_left_top_pixel.y+0.5));

    for (int i = max(0, left_top_pixel.x); i < min(width, right_bottom_pixel.x); i++) {
        for (int j = max(0,left_top_pixel.y); j < min(height, right_bottom_pixel.y); j++) {
            data[j * width + i] = make_uchar4(255, 0, 0, 255);
        }
    }
}


__global__ void drawBackground(uchar4* data, float* f_val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;
    int i, j;
    for (i = idx; i < width; i += offsetx) {
        for (j = idy; j < height; j += offsety) {
            float f = ((f_val[j * width + i] - dev_minFvalue) / (dev_maxFvalue - dev_minFvalue));
            data[j * width + i] = make_uchar4(int(f * 255),  int((1-f) * 255), 0, 255);
        }
    }
}


__global__ void updateVelocity(float2 *points, float2 *v, float2 *localBest, int n, float inertion, float a1, float a2) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int offset = blockDim.x*gridDim.x;

    for (int i = idx; i < n; i += offset) {
        float2 cur_force = make_float2(0, 0);
        for (int j = 0; j < n; j++) {
            if (j != i) {
                float x_dist = points[i].x - points[j].x;
                float y_dist = points[i].y - points[j].y;

                float g = sqrtf(x_dist*x_dist + y_dist*y_dist);
                cur_force.x += x_dist/powf(g, 4);
                cur_force.y += y_dist/powf(g, 4);
            }
        }
        hiprandState_t state;
        int MAX = 1000000;
        int seed = dev_seed;
        hiprand_init(seed, idx, 0, &state);
        float r1 = hiprand(&state) % MAX/float(MAX);
        float r2 = hiprand(&state) % MAX/float(MAX);
        v[i].x = v[i].x*inertion + (a1*r1*(dev_globalBest.x - points[i].x) + a2*r2*(localBest[i].x - points[i].x)+cur_force.x)*dt;
        r1 = hiprand(&state) % MAX/float(MAX);
        r2 = hiprand(&state) % MAX/float(MAX);
        v[i].y = v[i].y*inertion + (a1*r1*(dev_globalBest.y - points[i].y) + a2*r2*(localBest[i].y - points[i].y)+cur_force.y)*dt;
    }
}



void updateFMinMax() {
    thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(dev_Fvalue);
    float Fmin = thrust::min_element(dev_ptr, dev_ptr + width * height)[0];
    float Fmax = thrust::max_element(dev_ptr, dev_ptr + width * height)[0];
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_minFvalue), &Fmin, sizeof(float)));
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_maxFvalue), &Fmax, sizeof(float)));
}

void updateCenter(float2* dev_points) {
    float2 sum = thrust::reduce(thrust::device, dev_points, dev_points + points_num, make_float2(0, 0), thrust::plus<float2>());
    CSC(hipGetLastError());

    float avg_points_coord_influence_coef = 0.01;
    xCenter = xCenter + (sum.x / points_num - xCenter)*avg_points_coord_influence_coef;
    yCenter = yCenter + (sum.y / points_num - yCenter)*avg_points_coord_influence_coef;

    CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_xCenter), &xCenter, sizeof(float)));
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_yCenter), &yCenter, sizeof(float)));
}


__global__ void updateLocalMin(float2* points, float2* loc, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;
    for (int i = idx; i < n; i += offset) {
        if (Rozenbrock(points[i]) < Rozenbrock(loc[i])) {
            loc[i].x = points[i].x;
            loc[i].y = points[i].y;
        }
    }
}

void updateGlobalMin(float2* localBest) {
    thrust::device_ptr<float2> arr_ptr = thrust::device_pointer_cast(localBest);

    thrust::device_ptr<float2> min_dptr = thrust::min_element(thrust::device, arr_ptr, arr_ptr + points_num, cmp());

    float2* min_ptr = thrust::raw_pointer_cast(min_dptr);
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_globalBest), min_ptr, sizeof(float2)));
}

void drawPicture(uchar4* dev_data) {
    drawBackground<<<dim3(32, 32), dim3(32, 32)>>>(dev_data, dev_Fvalue);
    CSC(hipGetLastError());
    drawPoints<<<32, 32>>>(dev_data, dev_points, points_num);
    CSC(hipGetLastError()); 
}

struct hipGraphicsResource *res;


void update() {
    static float fps;
    static bool is_new_frame_counter = true;
    static int start;

    if (is_new_frame_counter) {
        start = clock();
        is_new_frame_counter = false;
    }
    uchar4* dev_data;
    size_t size;
    CSC(hipGraphicsMapResources(1, &res, 0));
    CSC(hipGraphicsResourceGetMappedPointer((void**)&dev_data, &size, res));

    updateGlobalMin(dev_localBest);
    updateVelocity<<<32, 32>>>(dev_points, dev_velocity, dev_localBest, points_num, inertion, a1, a2);
    updateCoord<<<32, 32>>>(dev_points, dev_velocity, points_num);
    updateCenter(dev_points);

    updateLocalMin<<<32, 32>>>(dev_points, dev_localBest, points_num);

    calculateF <<<dim3(32,32), dim3(32, 32)>>>(dev_Fvalue);
    updateFMinMax();
	
    drawPicture(dev_data);

    CSC(hipGraphicsUnmapResources(1, &res, 0));
    glutPostRedisplay();

    int finish = clock();
    if ((float)(finish - start)/CLOCKS_PER_SEC >= 1.0) {
        err = get_mean_dist_to_fmin_pos();
        printf("FPS: %.3f\n", fps);
        printf("Error (mean distination to fmin position): %.3f\n", err);
        is_new_frame_counter = true;
        fps = 0;
    }
    else
        fps++;
}

void display() {
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawPixels(width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);
    glutSwapBuffers();
}

void special_keys(int key, int x, int y) {
    float val = 1;
    if (key == GLUT_KEY_LEFT) {
        xCenter -= val;
    }
    else if (key == GLUT_KEY_RIGHT) {
        xCenter += val;
    }
    else if (key == GLUT_KEY_UP) {
        yCenter += val;
    }
    else if (key == GLUT_KEY_DOWN) {
        yCenter -= val;
    }

    CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_xCenter), &xCenter, sizeof(float)));
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_yCenter), &yCenter, sizeof(float)));
}

void keys(unsigned char key, int x, int y) {
    if (key == '-') {
        xScale *= 1.1;
        yScale = xScale * height / width;
        CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_yScale), &yScale, sizeof(float)));
        CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_xScale), &xScale, sizeof(float)));
    } else if (key == '+') {
        xScale /= 1.1;
        yScale = xScale * height / width;
        CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_yScale), &yScale, sizeof(float)));
        CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_xScale), &xScale, sizeof(float)));
    }
    if (key == 'q') {
        CSC(hipGraphicsUnregisterResource(res));
        glBindBuffer(1, vbo);
        glDeleteBuffers(1, &vbo);
        exit(0);
    }
}

void print_wrong_args_msg() {
    printf("Wrong arguments. They should be in this order: [-w] [-a1 -a2]\n");
    exit(0);
}

int main(int argc, char** argv) {
    srand(time(NULL));
    if (argc == 7) {
        if (strcmp(argv[1], "-w") == 0 && strcmp(argv[3], "-a1") == 0 && strcmp(argv[5], "-a2") == 0) {
            inertion = atof(argv[2]);
            a1 = atof(argv[4]);
            a2 = atof(argv[6]);
        } else
            print_wrong_args_msg();
    } else if (argc == 5) {
        if (strcmp(argv[1], "-a1") == 0 && strcmp(argv[3], "-a2") == 0) {

            a1 = atof(argv[2]);
            a2 = atof(argv[4]);
        } else
            print_wrong_args_msg();
    } else if (argc == 3) {
        if (strcmp(argv[1], "-w") == 0) {
            inertion = atof(argv[2]);
        } else
            print_wrong_args_msg();
    } else if (argc != 1)
        print_wrong_args_msg();

    CSC(hipMalloc(&dev_points, points_num * sizeof(float2)));
    CSC(hipMalloc(&dev_Fvalue, width * height * sizeof(float)));
    CSC(hipMalloc(&dev_localBest, points_num * sizeof(float2)));
    CSC(hipMalloc(&dev_dist, points_num * sizeof(float)));

    CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_xScale), &xScale, sizeof(float)));
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_yScale), &yScale, sizeof(float)));
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_xCenter), &xCenter, sizeof(float)));
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_yCenter), &yCenter, sizeof(float)));
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_minFvalue), &minFvalue, sizeof(float)));
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(dev_seed), &seed, sizeof(int)));


    float2* temp = (float2*)malloc(points_num * sizeof(float2));
    for (int i = 0; i < points_num; i++) {
        temp[i].x = xScale * (2 * (rand() / float(RAND_MAX)) - 1) + xCenter;
        temp[i].y = yScale * (2 * (rand() / float(RAND_MAX)) - 1) + yCenter;
    }
    CSC(hipMemcpy(dev_points, temp, points_num * sizeof(float2), hipMemcpyHostToDevice));
    CSC(hipMemcpy(dev_localBest, temp, points_num * sizeof(float2), hipMemcpyHostToDevice));

    CSC(hipMalloc(&dev_velocity, points_num * sizeof(float2)));
    for (int i = 0; i < points_num; i++) {
        temp[i].x = 10*xScale * (2 * (rand() / float(RAND_MAX)) - 1);
        temp[i].y = 10*xScale * (2 * (rand() / float(RAND_MAX)) - 1);
    }
    CSC(hipMemcpy(dev_velocity, temp, points_num * sizeof(float2), hipMemcpyHostToDevice));
    free(temp);

    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
    glutInitWindowSize(width, height);
    glutCreateWindow("Zabelin KP");

    glutIdleFunc(update);
    glutDisplayFunc(display);
    glutKeyboardFunc(keys);
    glutSpecialFunc(special_keys);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluOrtho2D(0.0, (GLfloat) width, 0.0, (GLfloat) height);

    glewInit();

    GLuint vbo;
    glGenBuffers(1, &vbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, vbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, width * height * sizeof(uchar4), NULL, GL_DYNAMIC_DRAW);

    CSC(hipGraphicsGLRegisterBuffer(&res, vbo, cudaGraphicsMapFlagsWriteDiscard));

    glutMainLoop();

    CSC(hipGraphicsUnregisterResource(res));

    glBindBuffer(1, vbo);
    glDeleteBuffers(1, &vbo);
    return 0;
}
